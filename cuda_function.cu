#include "hip/hip_runtime.h"
#include "cuda_function.h"

__device__  __host__ float f(int *pic, int *obj,int pic_dim,int obj_dim,int pic_start_row,int pic_start_col){
    float sum=0;
    for(int i=0;i<obj_dim;i++){
        for(int j=0;j<obj_dim;j++){
            float pic_val=pic[(pic_start_row+i)*pic_dim+pic_start_col+j];
            float obj_val=obj[i*obj_dim+j];
            sum+=fabs(pic_val-obj_val)/pic_val;
        }
    }
    return sum;
}

__global__ void kernel(int *pic, int *obj, float* result,int pic_dim,int obj_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int pic_row=i/pic_dim;
    int pic_col=i%pic_dim;
    result[i]=f(pic,obj,pic_dim,obj_dim,pic_row,pic_col);
}

void transfer_2D_mat_to_1D(int *one_dim_mat,int **mat_h,int N){
    for(int i=0;i<N;i++)
        for(int j=0;j<N;j++)
            one_dim_mat[i*N+j]=mat_h[i][j];
}

void check_cuda_allocation(hipError_t err,int type){
    if (err != hipSuccess) {
        if(type==1)
            fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        if(type==2)
            fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        if(type==3)
            fprintf(stderr, "Failed to copy data from device to host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
int computeOnGPU(manager *my_data, int object_index){
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int pic_dim=my_data->pictures[0].dim;
    int *pic;
    int temp_1D_pic[pic_dim*pic_dim];
    const size_t pic_size = sizeof(int) * size_t(pic_dim*pic_dim);
    err =hipMalloc((void **)&pic,  pic_size);
    check_cuda_allocation(err,1);

    transfer_2D_mat_to_1D(temp_1D_pic,my_data->pictures[0].mat,pic_dim);
    err = hipMemcpy(pic, temp_1D_pic,  pic_size, hipMemcpyHostToDevice);
    check_cuda_allocation(err,2);
    

    int obj_dim=my_data->objects[object_index].dim;
    int *obj;
    int temp_1D_obj[obj_dim*obj_dim];
    const size_t obj_size = sizeof(int) * size_t(obj_dim*obj_dim);
    err =hipMalloc((void **)&obj,  obj_size);
    check_cuda_allocation(err,1);

    transfer_2D_mat_to_1D(temp_1D_obj,my_data->objects[object_index].mat,obj_dim);
    err = hipMemcpy(obj, temp_1D_obj,  obj_size, hipMemcpyHostToDevice);
    check_cuda_allocation(err,1);


    float *result;
    const size_t result_size = sizeof(float) * size_t(pic_dim*pic_dim);
    err =hipMalloc((void **)&result,  result_size);
    check_cuda_allocation(err,1);

    float *host_result;
    host_result=(float*) malloc(result_size);
    if (host_result == NULL){
        printf("Error allocating memory");
        exit(EXIT_FAILURE);
    }

    int bool_found_match=0;
    kernel<<<pic_dim-obj_dim+1, pic_dim-obj_dim+1>>>(pic, obj, result, pic_dim, obj_dim);
    
    err = hipMemcpy(host_result, result,  result_size, hipMemcpyDeviceToHost);//copy result arry to device
    check_cuda_allocation(err,3);
    
    for(int i=0;i<pic_dim*pic_dim;i++){
        if(host_result[i]<my_data->matching_value){
            bool_found_match=1;
            add_matching(my_data,my_data->pictures[0].ID,my_data->objects[object_index].ID,i/pic_dim,i%pic_dim);
            break;
        }
    }

    if (hipFree(obj) != hipSuccess || hipFree(pic) != hipSuccess|| hipFree(result) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    free(host_result);

    return bool_found_match;
}


